
#include <hip/hip_runtime.h>
//CUDA runtime for KD_nodes



/*Roadmap: 

			(We are here)
 			 |
			\ /
			 v
Core functionality -> Dynamic Parralelism Experimentation -> Linking to existing libnabo framework -> optimization -> finalization

/Optimization |= (Search key sorting, linearizing the KD tree, improving GPU caching for node heaps)/

EST: Probably a month? Unsure. I need to look through the rest of the SDK.*/
#define maximum_depth 22
#define dim_count 3
#define K_size 16
#ifndef FLOAT_MAX 
#define FLOAT_MAX 33554430.0f
#endif 
#define BLOCK_SIZE 32
#define max_rad 256
//If coordinates are within 5% of eachother when compared to their cluster maximimum, treat them as a single point. To be used later
#define max_error 0.05f

#define OFFSIDE 0
#define ONSIDE 1
#define POINT_STRIDE 3
struct point
{
	float data[dim_count];
};
struct heap_entry
{
	float value;
	unsigned char index;
};
struct stack_entry{
	size_t n;
	uint state;
};

__device__ float heapHeadValue(heap_entry* h)
{
	return h->value;
}

__device__ heap_entry* heapHeadReplace(heap_entry* h, const int index, const float value, const uint K)
{
	uint i = 0;
	for (; i < K - 1; ++i)
	{
		if (h[i + 1].value > value)
			h[i] = h[i + 1];
		else
			break;
	}
	h[i].value = value;
	h[i].index = index;
	return h;
}

__device__ heap_entry *heapInit(const uint K)
{
	heap_entry *h;
	for (uint i = 0; i < K; ++i)
		h[i].value = FLOAT_MAX;
	return h;
}
struct kd_node
{
	//Which dimension
	unsigned int dim;
	//At what value was this node split?
	int cutVal;
	//The index of the current node
	int index;
};


#define inx_size 12
struct /*__align__(inx_size)*/ indx
{
	//The points of the KD tree
	point *pts;
	//The linked nodes
	const kd_node *nodes;
};

//Just a utility function for converting an int equal to zero to one, and vice versa. Its not well optimized, but it was quick to write :P Would be better with bitshifts
__device__ int flip(int in)
{
	return abs(in - 1);
}
__device__ unsigned int childLeft(const unsigned int pos) { return 2*pos + 1; }
__device__ unsigned int childRight(const unsigned int pos) { return 2*pos + 2; }
struct maxAB
{
	float A,B;
	int indx_a, indx_b;
};

//Clamp the value to 1 or 0
__device__ static float intensity(float a)
{
	return fmax(1,fmin(0,fabs(a)));
}
struct heap
{
	heap_entry *entries;
	int current_count;
};
//If dynamic parrallelism is not available, default to compute model 3_2. Eg: The early 700 series
#ifndef CM3_5
#define CM3_2
#endif
//Used to see if we're within bounds and are ready to jump a node
__device__ unsigned int withinBounds(int cd, point q, point p, float heapHeadVal, float maxRad, float maxError)
{
	float diff = q.data[cd] -p.data[cd];
	float side2 = diff*diff;
	if ((side2 <= maxRad) &&
		(side2 * maxError < heapHeadVal))
	{ 
		return 1;
	}
	return 0;
}
//Used for warp devices. One if distance is greater than zero. Returns 0 or 1 
__device__ unsigned int nodeMinor(int cd, point q, point p)
{
	float diff = q.data[cd] -p.data[cd];
	return (unsigned int)intensity(diff); 
	
}
//Implementation details: http://on-demand.gputechconf.com/gtc/2012/presentations/S0079-Warped-Parallel-Nearest-Neighbor-Searches-Using-KD-Trees.pdf
__device__ void recursive_warp_search(const indx static_data, const point query_point,  unsigned int _Mask, heap *output, 
					uint *stackpointer, stack_entry *stack, stack_entry *s)
{
	//Go up one
	--stackpointer;
	const size_t n = s->n;
	const kd_node node = static_data.nodes[n];
	const int cd = node.cutVal;
	//Continue doesn't do anything anymore since we're in a __device__ function (Not __global__), and there is no while loop
	/*if (cd == -2)
		continue;*/
	const int index = node.index;
	point p = static_data.pts[index];
	// compute new distance and update if lower
	float dist = 0;
	for (uint i = 0; i < dim_count; ++i)
	{
		const float diff = query_point.data[i] - p.data[i];
		dist += diff * diff;
	}
	if ((dist <= max_rad) &&
		(dist < heapHeadValue(output->entries)) &&
		(dist > (float)max_error)){
		output->entries = heapHeadReplace(output->entries, index, dist, K_size);output->current_count++;}
		// look for recursion
	//Let the warp group decide which way we want to travel next
	_Mask = _Mask & __ballot(withinBounds(cd, query_point,p,heapHeadValue(output->entries), max_rad, max_error));
	
	//This is going to be a very large amount of nested if statements o.O Luckily nvcc cleans it up a lot :D I'll eventually make it look nicer here as well

	//Did we go through the first branch?
	bool check = false;
	//If the group decided to travel off side first, continue
	if(_Mask!=0)
	{
		check = true;
		stackpointer++;
		_Mask = _Mask & __ballot(nodeMinor(cd,query_point,p));
		//Make a copy of S incase we want to branch right later on
		stack_entry s_copy1 = *s;
		s->state = OFFSIDE;
		//Branch left first
		if(_Mask!=0)
		{
			s->n = childLeft(n);	
			//Make another copy of s to come back to late
			stack_entry s_copy2 = *s;
			_Mask = _Mask & __ballot(nodeMinor(cd,query_point,p)); 
			*s = stack[*stackpointer];
			s->state = ONSIDE;	
			//We're going to branch to the right	
			stackpointer++;		
			if(_Mask !=0)
			{
				s->n = childRight(n);
				//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
				recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack, s);
				*s = s_copy2;
				//If any threads want to go to the left			
				if(__any(flip(nodeMinor(cd,query_point,p))))
				{
					//If I did not want this decision
					if(!flip(nodeMinor(cd,query_point,p)))
					{ goto exit; }
					s->n = childLeft(n);
					//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
					recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack,s);
				}
			}
			//If the group would rather branch left second
			else
			{
				s->n = childLeft(n);
				//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
				recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack, s);
				*s = s_copy2;
				//If any threads want to go to the left			
				if(__any(flip(nodeMinor(cd,query_point,p))))
				{
					//If I did not want this decision
					if(!flip(nodeMinor(cd,query_point,p)))
					{ goto exit; }
					s->n = childRight(n);
					//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
					recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack,s);
				}				
			}	
			stackpointer++;
			//We just came out of that branch, so lets set where we currently are in the stack back to s
			*s = s_copy1;
			//If any threads wanted to go right instead of left, branch that way now. In a worst case scenario, half the threads from the group are now gone
			if(__any(flip(nodeMinor(cd,query_point,p))))
			{
				//If I did not want this decision
				if(!flip(nodeMinor(cd,query_point,p)))
				{ goto exit; }
				s->n = childRight(n);
			}
		}
		//Branch right first
		else if(_Mask== 0)
		{
			s->n = childRight(n);	
			//Make another copy of s to come back to late
			stack_entry s_copy2 = *s;
			_Mask = _Mask & __ballot(nodeMinor(cd,query_point,p)); 
			*s = stack[*stackpointer];
			s->state = ONSIDE;	
			//We're going to branch to the right	
			stackpointer++;		
			if(_Mask !=0)
			{
				s->n = childRight(n);
				//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
				recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack, s);
				*s = s_copy2;
				//If any threads want to go to the left			
				if(__any(flip(nodeMinor(cd,query_point,p))))
				{
					//If I did not want this decision
					if(!flip(nodeMinor(cd,query_point,p)))
					{ goto exit; }
					s->n = childLeft(n);
					//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
					recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack, s);
				}
			}
			//If the group would rather branch left second
			else
			{
				s->n = childLeft(n);
				//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
				recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack, s);
				*s = s_copy2;
				//If any threads want to go to the left			
				if(__any(flip(nodeMinor(cd,query_point,p))))
				{
					//If I did not want this decision
					if(!flip(nodeMinor(cd,query_point,p)))
					{ goto exit; }
					s->n = childRight(n);
					//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
					recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack, s);
				}				
			}	
			//We just came out of that branch, so lets set where we currently are in the stack back to s
			*s = s_copy1;
			//If any threads wanted to go right instead of left, branch that way now. In a worst case scenario, half the threads from the group are now gone
			if(__any(flip(nodeMinor(cd,query_point,p))))
			{
				s->n = childLeft(n);
			}
		}		
	}
	//We want to branch onside and not offside
	else if((_Mask== 0) || ((check == true) && (__any(flip(nodeMinor(cd,query_point,p))))))
	{
		//Make another copy of s to come back to late
		stack_entry s_copy2 = *s;
		_Mask = _Mask & __ballot(nodeMinor(cd,query_point,p)); 
		*s = stack[*stackpointer];
		s->state = ONSIDE;	
		//We're going to branch to the right	
		stackpointer++;		
		if(_Mask !=0)
		{
			s->n = childRight(n);
			//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
			recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack, s);
			*s = s_copy2;
			//If any threads want to go to the left			
			if(__any(flip(nodeMinor(cd,query_point,p))))
			{
				//If I did not want this decision
				if(!flip(nodeMinor(cd,query_point,p)))
				{ goto exit; }
				s->n = childLeft(n);
				//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
				recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack, s);
			}
		}
		//If the group would rather branch left second
		else
		{
			s->n = childLeft(n);
			//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
			recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack, s);
			*s = s_copy2;
			//If any threads want to go to the left			
			if(__any(flip(nodeMinor(cd,query_point,p))))
			{
				//If I did not want this decision
				if(!flip(nodeMinor(cd,query_point,p)))
				{ goto exit; }				
				s->n = childRight(n);
				//Execute branch down. Eventually this will be doing through dynamic parrallelism instead of direct recursion				
				recursive_warp_search(static_data, query_point, _Mask, output, stackpointer, stack, s);
			}				
		}	
	}
exit:
	return;
	//TODO: Sort
	
} 
/*Kernel is to be executed as 32x1
indx is pre malloced and copied to the GPU to avoid memory bottlenecks. Query points is copied per iteration.
Uses a warped ballot system. Preferable for clustered points that are closely together.
Make sure the thread group size is equal to the size of the cluster & is a multiple of 32*/
__global__ void clustered_search(indx static_data, const point *query_points, int *indices,  heap *ret, int query_amt)
{
	stack_entry stack[maximum_depth];
	//Global thread number
	int thread_num = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	heap myHeap;
	myHeap.entries = heapInit(K_size);
	myHeap.current_count = 0;
	//Start at root node
	stack_entry* s = stack;
	uint *startpos;
	*startpos = 0;
	recursive_warp_search(static_data, query_points[thread_num], 1, &myHeap, startpos,s,stack);
	ret[thread_num] = myHeap;
} 

